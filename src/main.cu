#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include ""
#include "Draw.hpp"
#include "Startup.hpp"
#include "NodeGeneration.hpp"
#include <iostream>
#include <cstdlib>
#include "forceDirectedPlacement.hpp"

#define gpuErrchk(ans) gpuAssert((ans), __FILE__, __LINE__);
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code == hipSuccess) return;
	fprintf(stderr, "Error: %s %s Ln: %d\n", hipGetErrorString(code), file, line);
	if (abort) exit(code);
}

int main(int argc, char* argv[])
{
	// Initialisation
	const auto windowWidth = 800;
	const auto windowHeight = 600;
	auto windowSize = Vector2(windowWidth, windowHeight);
	std::vector<Node*> nodes;

	// Handling parameters
	auto args = handleArgs(argc, argv);
	nodes = (args.fileName == nullptr) ?
		getNodesRandom(args.numNodes, args.seed) : handleFile(args.fileName);

	// Force directed placement
	const auto SPREADOFFSET = 1.0f; // Known as the C value
	auto spread = SPREADOFFSET * sqrtf(1.0f * windowWidth * windowHeight / args.numNodes);
	auto scale = windowWidth + windowHeight;
	auto fdp = FdpContext(nodes, windowSize, scale, spread, args.iterations);
	forceDirectedPlacement(fdp, args);
	Node::printNodesAndConnections(nodes);

	// Setting up window
	auto draw = new Draw((char*)"CPU Implementation", windowWidth, windowHeight);
	draw->draw(nodes);

	// Cleanup
	delete draw;
	nodes.clear();
	return 0;
}