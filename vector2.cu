#include "hip/hip_runtime.h"
#include "vector2.cuh"
#include <cmath>

template <class T>
inline void Vector2<T>::reset(void)
{
    this->x = 0;
    this->y = 0;
};

template <class T>
inline float Vector2<T>::distance(Vector2<float>& node)
{
    return sqrtf(powf(this->y - node.y, 2) + powf(this->x - node.x, 2));
};

template <typename T>
inline Vector2<T> operator-(const Vector2<T>& right)
{
    return Vector2<T>(-right.x, -right.y);
}

template <typename T>
inline Vector2<T>& operator+=(Vector2<T>& left, const Vector2<T>& right)
{
    left.x += right.x;
    left.y += right.y;

    return left;
}

template <typename T>
inline Vector2<T>& operator-=(Vector2<T>& left, const Vector2<T>& right)
{
    left.x -= right.x;
    left.y -= right.y;

    return left;
}

template <typename T>
inline Vector2<T> operator+(const Vector2<T>& left, const Vector2<T>& right)
{
    return Vector2<T>(left.x + right.x, left.y + right.y);
}

template <typename T>
inline Vector2<T> operator-(const Vector2<T>& left, const Vector2<T>& right)
{
    return Vector2<T>(left.x - right.x, left.y - right.y);
}

template <typename T>
inline Vector2<T> operator*(const Vector2<T>& left, T right)
{
    return Vector2<T>(left.x * right, left.y * right);
}

template <typename T>
inline Vector2<T> operator*(T left, const Vector2<T>& right)
{
    return Vector2<T>(right.x * left, right.y * left);
}

template <typename T>
inline Vector2<T>& operator*=(Vector2<T>& left, T right)
{
    left.x *= right;
    left.y *= right;

    return left;
}

template <typename T>
inline Vector2<T> operator/(const Vector2<T>& left, T right)
{
    return Vector2<T>(left.x / right, left.y / right);
}

template <typename T>
inline Vector2<T>& operator/=(Vector2<T>& left, T right)
{
    left.x /= right;
    left.y /= right;

    return left;
}

template <typename T>
inline bool operator==(const Vector2<T>& left, const Vector2<T>& right)
{
    return (left.x == right.x) && (left.y == right.y);
}

template <typename T>
inline bool operator!=(const Vector2<T>& left, const Vector2<T>& right)
{
    return (left.x != right.x) || (left.y != right.y);
}