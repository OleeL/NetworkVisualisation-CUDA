﻿#include "graph.cuh"
#include "vector2.cuh"
#include <iostream>

Graph::Graph() {
	this->nodes = nullptr;
	this->edges = nullptr;
	this->displacement = nullptr;
	this->distinctEdges = nullptr;
	this->connectionIndex = nullptr;
	this->numberOfNodes = 0;
	this->numberOfEdges = 0;
};

Graph::Graph(Vector2f* nodes,
	Vector2f* displacement,
	Vector2i* edges,
	Vector2i* distinctEdges,
	unsigned int* connectionIndex,
	unsigned int numberOfNodes,
	unsigned int numberOfEdges) :
	nodes(nodes),
	displacement(displacement),
	edges(edges),
	distinctEdges(distinctEdges),
	connectionIndex(connectionIndex),
	numberOfNodes(numberOfNodes),
	numberOfEdges(numberOfEdges)
{};

void Graph::printNodes() {
	for (unsigned int i = 0; i < this->numberOfNodes; ++i)
	{
		std::cout
			<< i
			<< ":\t"
			<< this->nodes[i].x
			<< ", "
			<< this->nodes[i].y
			<< std::endl;
	}
}

inline void Graph::printNodesAndConnections(Graph& graph) {
	for (unsigned int i = 0; i < graph.numberOfEdges * 2; ++i) {
		std::cout
			<< graph.edges[i].x
			<< " connected to "
			<< graph.edges[i].y
			<< std::endl;
	}
}
