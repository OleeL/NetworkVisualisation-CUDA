﻿#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <cstdlib>
#include "draw.cuh"
#include "graph.cuh"
#include "startup.cuh"
#include "nodeGeneration.cuh"
#include "forceDirectedPlacement.cuh"

int main(int argc, char* argv[])
{
	// Handling parameters
	auto args = handleArgs(argc, argv);

	// Initialisation
	auto graph = handleFile(args.fileName);

	forceDirectedPlacement(args, graph);

	// Setting up window
	auto draw = new Draw((char*) "GPU Implementation", args.windowSize.x, args.windowSize.y);
	draw->draw(graph);

	// Cleanup
	delete draw;
	return 0;
}