﻿#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <cstdlib>
#include "startup.cu"
#include "nodeGeneration.cu"
#include "forceDirectedPlacement.cu"
#include "graph.cu"

int main(int argc, char* argv[])
{
	// Handling parameters
	ParamLaunch* args = handleArgs(argc, argv);

	// Initialisation
	Graph* graph = handleFile(args->fileName);

	// Running algorithm
	forceDirectedPlacement(args, graph);

	graph->destroy();
	free(args);
	free(graph);
	return 0;
}
