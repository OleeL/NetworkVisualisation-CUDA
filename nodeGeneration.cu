#include "hip/hip_runtime.h"
#include <algorithm>    // std::random_shuffle std::sort
#include <fstream>
#include <sstream>
#include <iostream>
#include "nodeGeneration.cuh"
#include "graph.cuh"
#include "vector2.cuh"

inline void goToLine(std::ifstream& file, int line)
{
	std::string s;
	file.clear();
	file.seekg(0);
	for (auto i = 0; i < line; i++)
		std::getline(file, s);
};

// Before edges looked like:
// 4 1
// 9 2
// 5 3
// ...
// Now, they look like
// 0 1
// 0 2
// 0 3
// 1 0
// 1 2
// ...
bool order(float2 &a, float2 &b)
{
	if (a.x < b.x) return true;
	return (a.y < b.y && a.x == b.x);
}

Graph handleFile(char* fileName) {
	unsigned int lines, nNodes;

	std::ifstream file;
	file.open(fileName);
	file >> nNodes >> lines;

	auto nodes = new float2[nNodes];
	auto displacement = new float2[nNodes]();
	auto distinctEdges = new int2[lines];
	auto adjacencyMatrix = new int[int(nNodes * nNodes)]();
	auto v = int2();
	auto f = float2();

	// Looping through all nodes
	for (unsigned int i = 0; i < nNodes; ++i)
	{
		auto float2();
		f.x = static_cast<float>(rand()) / RAND_MAX - 0.5f;
		f.y = static_cast<float>(rand()) / RAND_MAX - 0.5f;
		nodes[i] = f;
	}
	// Looping through all distinct edges
	for (unsigned int i = 0; i < lines; ++i)
	{
		file >> v.x >> v.y;
		distinctEdges[i] = v;
		adjacencyMatrix[v.x * nNodes + v.y] = 1;
		adjacencyMatrix[v.y * nNodes + v.x] = 1;
	}
	file.close();

	return Graph(nodes, displacement, distinctEdges, adjacencyMatrix, nNodes, lines);
}